
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h> 
#include <stdlib.h>
#include <bits/stdc++.h>
#include <random>
#include <chrono>
#include <ctime>
#include <xmmintrin.h>
#include<sys/time.h>
#include<time.h>
#include <emmintrin.h>

using namespace std;

#define N0  784
#define N1  1000
#define N2  500
#define N3 4
#define A  1.7159
#define B  0.6666


int label_4[24754];
int label_4_pos[24754];

double *IN, *W0, *B1, *H1S, *H1O, *W1, *B2, *H2S, *H2O, *W2, *B3, *OS, *OO;
double *dE_OO, *dOO_OS, *dE_OS, *dE_B3, *dE_W2, *dE_H2O, *dH2O_H2S, *dE_H2S, *dE_B2, *dE_W1, *dE_H1O, *dH1O_H1S, *dE_H1S, *dE_B1, *dE_W0;
double *B1_dev;



double small_X[24754][784];
double small_y[24754][4];
double err;
double rate = 0.005;

int counts[N3];
int n_rows=28;
int n_cols=28;
int  image[60000][28][28];
int  label[60000];
int  predicted_label[60000];

double X_data[60000][784];
double  Y_label[60000][10];

int cnt_Z=0;

int reverseInt (int i)
{
    unsigned char c1, c2, c3, c4;

    c1 = i & 255;
    c2 = (i >> 8) & 255;
    c3 = (i >> 16) & 255;
    c4 = (i >> 24) & 255;

    return ((int)c1 << 24) + ((int)c2 << 16) + ((int)c3 << 8) + c4;
}

double sigmoid(double x)
{
    //cout<<x<<endl;
    double xx = A*tanh(B*x);
	return xx;
}



void training_image(){
    ifstream file ("train-images.idx3-ubyte");
    if (file.is_open()){
        int magic_number=0;
        int number_of_images=0;
        file.read((char*)&magic_number,sizeof(magic_number));
        magic_number= reverseInt(magic_number);
        file.read((char*)&number_of_images,sizeof(number_of_images));
        number_of_images= reverseInt(number_of_images);
        file.read((char*)&n_rows,sizeof(n_rows));
        n_rows= reverseInt(n_rows);
        file.read((char*)&n_cols,sizeof(n_cols));
        n_cols= reverseInt(n_cols);

        for(int i=0;i<number_of_images;++i)
        {
            for(int r=0;r<n_rows;++r)
            {
                for(int c=0;c<n_cols;++c)
                {
                    unsigned char temp=0;
                    file.read((char*)&temp,sizeof(temp));
                    image[i][r][c] = temp;
                }
            }

        }  
        //display_image_by_id(7000);
    }
    else{
        cout<<"Unable to openfile \n";
        exit(0);
    }
}

void training_label(){
    int number_of_images=0;
    ifstream file ("train-labels.idx1-ubyte");
    if (file.is_open())
    {
        int magic_number=0;        
        file.read((char*)&magic_number,sizeof(magic_number));
        magic_number= reverseInt(magic_number);
        file.read((char*)&number_of_images,sizeof(number_of_images));
        number_of_images= reverseInt(number_of_images);
        for(int i=0;i<number_of_images;++i)
        {
            unsigned char temp=0;
            file.read((char*)&temp,sizeof(temp));
            label[i]= temp;
            counts[temp] += 1;  
        }  
        //display_label_by_id(7000);
    }
}


void create_small_dataset(){
	int c = 0;
	for (int i = 0; i<60000; i++){
		if (label[i] < 4){
			label_4[c] = label[i];
			label_4_pos[c] = i;
			c+=1;
		}
	}
}

int counter = 0;

void forward(double *input)
{

        for (int i = 0; i<N0; i++) 
		
		{

			IN[i]   = input[i];
		}


		

        // compute the weighted sum HS in the hidden layer
        for (int i=0; i<N1; i++) 
        {

			H1S[i] = B1[i];
		}


        for (int i=0; i<N1; i++) {
                for (int j=0; j<N0; j++){
                        H1S[i] += IN[j]*W0[N1*j+i];
            //cout<<W0[j][i]<<endl;
            }
        }

//matched so far

 

 //        // Comput the output of the hidden layer, HO[N1];

        for (int i=0; i<N1; i++) 
        {

			H1O[i] = sigmoid(H1S[i]);

		}


	//compute weighted sum of H2
	for (int i=0; i<N2; i++) {
		H2S[i] = B2[i];
        
	}

        for (int i=0; i<N2; i++) {
                for (int j=0; j<N1; j++){
                        H2S[i] += H1O[j]*W1[N2*j+i];
            //cout<<W0[j][i]<<endl;
            }
        }


	//compute the output of H2

	for (int i=0; i<N2; i++) {
		H2O[i] = sigmoid(H2S[i]);
        //cout<<HS[i]<<"   "<<HO[i]<<endl;
	}

        // compute the weighted sum  in the output layer
    for (int i=0; i<N3; i++) {
		OS[i] = B3[i];
	}


    for (int i=0; i<N3; i++) {
		for (int j=0; j<N2; j++)
			OS[i] += H2O[j]*W2[N3*j+i];
	}

//matched********

        // Comput the output of the output layer, OO[N2];

        for (int i=0; i<N3; i++) {
		OO[i] = sigmoid(OS[i]);
        
	}

		cout << "OO: ";
	for(int i = 0; i<4; i++)
		cout << OO[i] << " ";
	cout << endl;

	counter++;
	if (counter>10)
		exit(0);
	// cout << "forward complete" << endl;
	// exit(0);
}


void backward(double *O, double *Y)
{
        // compute error
	err = 0.0;
        for (int i=0; i<N3; i++) 
		err += (O[i] - Y[i])*(O[i]-Y[i]);
	err = err / N3;

        // compute dE_OO
        for (int i=0; i<N3; i++) 
		dE_OO[i] = (O[i] - Y[i])*2.0/N3;

        // compute dOO_OS = OO dot (1-OO)
        for (int i=0; i<N3; i++)
		dOO_OS[i] = A*B*(1- ((OO[i]/A) * (OO[i]/A)));

        // compute dE_OS = dE_OO dot dOO_OS
        for (int i=0; i<N3; i++)
		dE_OS[i] = dE_OO[i] * dOO_OS[i];

        // compute dE_B3 = dE_OS
        for (int i=0; i<N3; i++)
		dE_B3[i] = dE_OS[i];



        // compute dE_W2
        for (int i=0; i<N2; i++)
		for (int j = 0; j<N3; j++) 
			dE_W2[i*N3+j] = dE_OS[j]*H2O[i];


//matched till here


	// 	//last layer done******************

	// compute dE_H2O
	 for (int i=0; i<N2; i++) {
	 	dE_H2O[i] = 0;
	 	for (int j = 0; j<N3; j++)
	 		dE_H2O[i] += dE_OS[j]*W2[i*N3+j];
	}

 //        // compute dH2O_H2S = H2O dot (1-H2O)
        for (int i=0; i<N2; i++)
         {
         	dH2O_H2S[i] = A*B*(1- ((H2O[i]/A) * (H2O[i]/A)));
         }
 
  // compute dE_H2S = dE_H2O dot dH2O_H2S
         for (int i=0; i<N2; i++)
		
 	{
 		 dE_H2S[i] = dE_H2O[i] * dH2O_H2S[i];

	}


 //        // compute dE_B2 = dE_H2S
       for (int i=0; i<N2; i++)
	 	dE_B2[i] = dE_H2S[i];

      // compute dE_W1
	  for (int i=0; i<N1; i++)
	 	for (int j = 0; j<N2; j++) 
	 		dE_W1[i*N2+j] = dE_H2S[j]*H1O[i];



//matched till here

	for (int  i=0; i<N1; i++) {
		dE_H1O[i] = 0;
		for (int j = 0; j<N2; j++)
		{
			dE_H1O[i] += dE_H2S[j]*W1[i*N2+j];
		}

	}

        // compute dH1O_H1S = H1O dot (1-H1O)
        for (int i=0; i<N1; i++)
		dH1O_H1S[i] = A*B*(1- ((H1O[i]/A) * (H1O[i]/A)));


        // compute dE_H1S = dE_H1O dot dH1O_H1S
        for (int i=0; i<N1; i++)
		
		{
			 dE_H1S[i] = dE_H1O[i] * dH1O_H1S[i];
			//_mm_storeu_pd(&dE_H1S[i],_mm_mul_pd(_mm_load_pd(&dE_H1O[i]),_mm_load_pd(&dH1O_H1S[i])));
		}
        // compute dE_B1 = dE_H1S
        for (int i=0; i<N1; i++)
		dE_B1[i] = dE_H1S[i];



        // compute dE_W0
        for (int i=0; i<N0; i++)
		for (int j = 0; j<N1; j++) 
			dE_W0[i*N1+j] = dE_H1S[j]*IN[i];




	// 	//******************************
	// /*
	// cout << "err = " << err << "\n";
	// print_1d(IN, N0, "IN");
	// print_1d(dE_OO, N2, "dE_OO");
	// print_1d(dOO_OS, N2, "dOO_OS");
	// print_1d(OO, N2, "OO");
	// print_1d(dE_OS, N2, "dE_OS");
 //        print_1d(dE_B2, N2, "dE_B2");
 //        print_12(dE_W1, "dE_W1");
 //        print_1d(dE_B1, N1, "dE_B1");
 //        print_01(dE_W0, "dE_W0");
	// */

 //        // update W0, W1, B1, B2;

	// for (int i=0; i<N0; i++)
	// 	for (int j=0; j<N1; j++)
	// 		W0[i][j] = W0[i][j] - rate * dE_W0[i][j];
	for (int i = 0; i<N0*N1; i++)
		W0[i] = W0[i] - rate*dE_W0[i];

	for (int i=0; i<N1; i++)
		B1[i] = B1[i] - rate * dE_B1[i];


	// for (int i=0; i<N1; i++)
	// 	for (int j=0; j<N2; j++)
	// 		W1[i][j] = W1[i][j] - rate * dE_W1[i][j];
	for (int i = 0; i<N1*N2; i++)
		W1[i] = W1[i] - rate*dE_W1[i];

	for (int i=0; i<N2; i++)
		B2[i] = B2[i] - rate * dE_B2[i];

	// for (int i=0; i<N2; i++)
	// 	for (int j=0; j<N3; j++)
	// 		W2[i][j] = W2[i][j] - rate * dE_W2[i][j];
	for (int i = 0; i<N2*N3; i++)
		W0[i] = W0[i] - rate*dE_W0[i];

	for (int i=0; i<N3; i++)
		B3[i] = B3[i] - rate * dE_B3[i];

}


// 	for(int i =0; i<10; i++)
// 	{
// 		cout << B3[i] << " ";
// 	}
// 	cout << " B3" << endl;
// }  

int ii;

double cal_acc()
{
	int c = 0;
	int pred_label[5000];

	for(int i = 0; i<5000; i++)
	{
		double temp_in[N0];
		forward(&(small_X[i][0]));
		double max_temp = OO[0];
		int k = 0;
		for (int j = 1; j<N3; j++){
			if (max_temp<OO[j]){
				max_temp = OO[j];
				k = j;
			}
		}
		// cout << "original " << label[i] << " , predicted: " << k << endl;
		if (label_4[i] == k)
			c+=1;
	}
	cout << "counter: " << c << endl;
	double acc = (c/5000.0) * 100.0;

	return acc;
}

void train(int iter)
{
	for (int i = 0; i< iter; i++) {
		//int ii = random () % 4;
		ii = i % 24754;
                //int ii= 3;
		forward(&(small_X[ii][0]));
		backward(OO, (&small_y[ii][0]));
				if (i % 10000 == 0) 
			{
                double acc;
                acc = cal_acc();
 //                MyFile.open("output.txt", std::ios_base::app);
                cout << "Iteration " << i << ": err =" << err << " acc: " << acc << "\n";
 //                string str;

 //                str = "At iteration "+ to_string(i)+": err = "+to_string(err)+", Y= "+to_string(label_4[ii])+"\n";
 //                MyFile << str;
 //                print_val(ii);
 //                MyFile.close();
                

 //            }

	// 	// break;
	// }


			}

}
}
		



int main()
{
	training_image();
    training_label();
    create_small_dataset();


    IN = (double*)malloc(N0*sizeof(double));
	W0 = (double*)malloc(N0*N1*sizeof(double));
	B1 = (double*)malloc(N1*sizeof(double));
	H1S = (double*)malloc(N1*sizeof(double));
	H1O = (double*)malloc(N1*sizeof(double));

	W1 = (double*)malloc(N1*N2*sizeof(double));
	B2 = (double*)malloc(N2*sizeof(double));
	H2S = (double*)malloc(N2*sizeof(double));
	H2O = (double*)malloc(N2*sizeof(double));

	W2 = (double*)malloc(N2*N3*sizeof(double));
	B3 = (double*)malloc(N3*sizeof(double));
	OS = (double*)malloc(N3*sizeof(double));
	OO = (double*)malloc(N3*sizeof(double));


	dE_OO = (double*)malloc(N3*sizeof(double));
	dOO_OS = (double*)malloc(N3*sizeof(double));
	dE_OS = (double*)malloc(N3*sizeof(double));
	dE_B3 = (double*)malloc(N3*sizeof(double));
	dE_W2 = (double*)malloc(N2*N3*sizeof(double));

	dE_H2O = (double*)malloc(N2*sizeof(double));
	dH2O_H2S = (double*)malloc(N2*sizeof(double));
	dE_H2S = (double*)malloc(N2*sizeof(double));
	dE_B2 = (double*)malloc(N2*sizeof(double));
	dE_W1 = (double*)malloc(N1*N2*sizeof(double));

	dE_H1O = (double*)malloc(N1*sizeof(double));
	dH1O_H1S = (double*)malloc(N1*sizeof(double));
	dE_H1S = (double*)malloc(N1*sizeof(double));
	dE_B1 = (double*)malloc(N1*sizeof(double));
	dE_W0 = (double*)malloc(N0*N1*sizeof(double));





	for (int i=0; i<60000;i++){
	    for (int j=0; j<28;j++){
	        for(int k=0;k<28;k++){
	            X_data[i][k+j*28]=(image[i][j][k]/127.5)-1;
	        }
	    }        
	} 

	for(int i=0; i<60000; i++){
	    for (int j =0; j<10; j++){
	        if (j==label[i]){
	            Y_label[i][j]=1.0;
	        }
	        else{
	            Y_label[i][j]=-1.0;
	        }
	    }             
	}

		for(int i = 0; i<24754; i++){
			int temp = label_4_pos[i];

			for (int j = 0; j<784; j++){
				small_X[i][j] = X_data[temp][j];
			}
		}

		for (int i = 0; i<24754; i++){
			int temp = label_4_pos[i];
			for (int j = 0; j<N3; j++){
				small_y[i][j] = Y_label[temp][j];

			}
		}


	// randomize weights
    for (int i = 0; i<N1; i++)
		B1[i] = random()*1.0/RAND_MAX/100;

  //   for (int i = 0; i<N0; i++)
		// for (int j = 0; j<N1; j++)
		// 	W0[i][j] = random()*1.0/RAND_MAX/100;
	for (int i =0; i<N0*N1; i++)
		W0[i] = random()*1.0/RAND_MAX/100;
    
    for (int i = 0; i<N2; i++)
		B2[i] = random()*1.0/RAND_MAX/100;
    
    for (int i = 0; i<N1*N2; i++)
		W1[i] = random()*1.0/RAND_MAX/100;

	for (int i = 0; i<N3; i++)
		B3[i] = random()*1.0/RAND_MAX/100;
    
    for (int i = 0; i<N2*N3; i++)
		W2[i] = random()*1.0/RAND_MAX/100;


	hipMalloc( &B1_dev, N1*sizeof(double) );
	hipMemcpy( B1_dev, B1, N1*sizeof(double), hipMemcpyHostToDevice );

	// for (int i =(N0-1)*1000; i<N0*1000+10; i++)
	// 	cout << W0[i] << " ";
	// cout << endl;

    train(100000000);





return 0;

}